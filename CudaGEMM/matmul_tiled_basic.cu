// Basic tiled matrix multiplication implementation

#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

const int N = 1 << 10;  // 1024 x 1024
const int TILE_SIZE = 32;

__global__ void matrixMul(const float *__restrict__ a, 
                         const float *__restrict__ b,
                         float *__restrict__ c) {
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    __shared__ float s_a[TILE_SIZE][TILE_SIZE];
    __shared__ float s_b[TILE_SIZE][TILE_SIZE];

    float sum = 0.0f;

    for (int m = 0; m < N; m += TILE_SIZE) {
        // Load tiles into shared memory
        if (row < N && m + threadIdx.x < N) {
            s_a[threadIdx.y][threadIdx.x] = a[row * N + m + threadIdx.x];
        } else {
            s_a[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (m + threadIdx.y < N && col < N) {
            s_b[threadIdx.y][threadIdx.x] = b[(m + threadIdx.y) * N + col];
        } else {
            s_b[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute partial dot product
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += s_a[threadIdx.y][k] * s_b[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write result
    if (row < N && col < N) {
        c[row * N + col] = sum;
    }
}

void verify_result(vector<float> &a, vector<float> &b, vector<float> &c) {
    const float epsilon = 1e-3;
    float max_diff = 0.0f;
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            float tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }
            float diff = std::abs(tmp - c[i * N + j]);
            max_diff = std::max(max_diff, diff);
            
            if (diff > epsilon) {
                cout << "Mismatch at (" << i << "," << j << "): " 
                     << "CPU=" << tmp << " GPU=" << c[i * N + j] 
                     << " diff=" << diff << "\n";
                assert(false);
            }
        }
    }
    cout << "Maximum difference: " << max_diff << "\n";
}

int main() {
    size_t bytes = N * N * sizeof(float);

    vector<float> h_a(N * N);
    vector<float> h_b(N * N);
    vector<float> h_c(N * N);

    generate(h_a.begin(), h_a.end(), []() { return (float)rand() / RAND_MAX; });
    generate(h_b.begin(), h_b.end(), []() { return (float)rand() / RAND_MAX; });

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks(N / TILE_SIZE, N / TILE_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warm-up run
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    // Measure performance
    hipEventRecord(start);
    matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    double total_ops = 2.0 * N * N * N;
    double gflops = (total_ops / (milliseconds / 1000.0)) / 1e9;

    cout << "Basic Tiled Implementation:\n";
    cout << "Matrix size: " << N << "x" << N << "\n";
    cout << "Execution time: " << milliseconds << " ms\n";
    cout << "GFLOPS: " << gflops << "\n";

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
    verify_result(h_a, h_b, h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
} 